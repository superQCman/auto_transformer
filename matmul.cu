#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <sys/time.h>
#include <fstream>
#include <iostream>
#include <string>
#include <iostream>
#include "apis_cu.h"
#include ""


__global__ void matmul_kernel(float* A, float* B, float* C, int M, int N, int K, int batch_size){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int batch = blockIdx.z * blockDim.z + threadIdx.z;

    if(row < M && col < N && batch < batch_size){
        float sum = 0.0f;
        for(int i = 0; i < K; i++){
            sum += A[batch * M * K + row * K + i] * B[batch * K * N + i * N + col];
        }
        C[row * N + col] = sum;
    }
} 


int main(int argc, char* argv[]){
    int srcX = atoi(argv[1]);
    int srcY = atoi(argv[2]);
    int dstX = atoi(argv[3]);
    int dstY = atoi(argv[4]);
    int block_size = atoi(argv[5]);

    int64_t batch_size = 0;
    int64_t M = 0;
    int64_t N = 0;
    int64_t K = 0;

    int64_t* send_size_d;
    int64_t send_size[4];
    hipMalloc((void**)&send_size_d, 4*sizeof(int64_t));

    receiveMessage( srcX, srcY, dstX, dstY, send_size_d, 4*sizeof(int64_t));
    hipMemcpy(send_size, send_size_d, 4*sizeof(int64_t), hipMemcpyDeviceToHost);
    batch_size = send_size[0];
    M = send_size[1];
    K = send_size[2];
    N = send_size[3];

    std::cout << "batch_size: " << batch_size << std::endl;
    std::cout << "M: " << M << std::endl;
    std::cout << "K: " << K << std::endl;
    std::cout << "N: " << N << std::endl;
    

    int64_t a_size = batch_size * M * K * sizeof(float);
    int64_t b_size = batch_size * K * N * sizeof(float);
    int64_t c_size = batch_size * M * N * sizeof(float);

    std::cout << "a_size: " << a_size << std::endl;
    std::cout << "b_size: " << b_size << std::endl;
    std::cout << "c_size: " << c_size << std::endl;

    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, a_size);
    hipMalloc((void**)&d_B, b_size);
    hipMalloc((void**)&d_C, c_size);

    // float* h_A = (float*)malloc(a_size);
    // float* h_B = (float*)malloc(b_size);
    // float* h_C = (float*)malloc(c_size);

    // for(int i = 0; i < batch_size; i++){
    //     for(int j = 0; j < M; j++){
    //         for(int k = 0; k < K; k++){
    //             h_A[i * M * K + j * K + k] = 1.0f;
    //         }
    //     }
    // }
    
    // for(int i = 0; i < batch_size; i++){
    //     for(int j = 0; j < K; j++){
    //         for(int k = 0; k < N; k++){
    //             h_B[i * K * N + j * N + k] = 1.0f;
    //         }
    //     }
    // }

    // hipMemcpy(d_A, h_A, a_size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B, h_B, b_size, hipMemcpyHostToDevice);
    receiveMessage(srcX, srcY, dstX, dstY, d_A, a_size);
    receiveMessage(srcX, srcY, dstX, dstY, d_B, b_size);

    dim3 block(block_size, block_size, block_size);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y, (batch_size + block.z - 1) / block.z);

    matmul_kernel<<<grid, block>>>(d_A, d_B, d_C, M, N, K, batch_size);
    // hipMemcpy(h_C, d_C, c_size, hipMemcpyDeviceToHost);
    // std::cout << "h_C: " << h_C[c_size - 1] << std::endl;

    sendMessage(dstX, dstY, srcX, srcY, d_C, c_size);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(send_size_d);

    // hipMemcpy(C, d_C, batch_size * M * N * sizeof(float), hipMemcpyDeviceToHost);
    
}